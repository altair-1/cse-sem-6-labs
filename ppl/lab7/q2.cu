#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__global__ void buildRS(char* s, char* rs, int len) {
    int i = threadIdx.x;
    int offset = 0;
    for (int j = 0; j < i; j++) {
        offset += len - j;
    }
    for (int j = 0; j < len - i; j++) {
        rs[offset + j] = s[j];
    }
}

int main() {
    char s[100];
    printf("enter input string S: ");
    scanf("%s", s);
    int len = strlen(s);

    int rs_len = len * (len + 1) / 2;  
    char* rs = (char*)malloc((rs_len + 1) * sizeof(char));  
    rs[rs_len] = '\0';

    char *d_s, *d_rs;
    hipMalloc((void**)&d_s, len * sizeof(char));
    hipMalloc((void**)&d_rs, rs_len * sizeof(char));

    hipMemcpy(d_s, s, len * sizeof(char), hipMemcpyHostToDevice);

    buildRS<<<1, len>>>(d_s, d_rs, len);
    hipDeviceSynchronize();

    hipMemcpy(rs, d_rs, rs_len * sizeof(char), hipMemcpyDeviceToHost);

    printf("output string RS: %s\n", rs);

    hipFree(d_s);
    hipFree(d_rs);
    free(rs);

    return 0;
}