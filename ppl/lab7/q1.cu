#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__device__ bool match_word(char* str, int idx, char* key, int keylen) {
    for (int i = 0; i < keylen; i++) {
        if (str[idx + i] != key[i]) {
            return false;
        }
    }
    return true;
}

__global__ void wordMatch(char* str, int len, char* key, int keylen, int *count) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < len - keylen + 1) {
        if (match_word(str, id, key, keylen)) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    char str[100];
    printf("enter a sentence: ");
    scanf("%[^\n]%*c", str);
    int len = strlen(str);
    
    char key[20];
    printf("enter a word from the sentence: ");
    scanf("%[^\n]%*c", key);
    int keylen = strlen(key);
    
    int count = 0;
    char *d_str, *d_key;
    int *d_count;
    
    hipMalloc((void**)&d_str, len * sizeof(char));
    hipMalloc((void**)&d_key, keylen * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));
    
    hipMemcpy(d_str, str, len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, keylen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    wordMatch<<<ceil((float)len/256), 256>>>(d_str, len, d_key, keylen, d_count);
    hipDeviceSynchronize();
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("occurrences of the word '%s' in the sentence: %d\n", key, count);

    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_count);
    return 0;
}