#include "hip/hip_runtime.h"

#include <stdio.h>

const int n = 1000;

__global__ void vector_addition(int *a, int *b, int *c) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid < n) c[tid] = a[tid] + b[tid];
}

int main() {
    int *d_a, *d_b, *d_c; // device copies
    int size = sizeof(int);
    
    hipMalloc((void**)&d_a, n * size);
    hipMalloc((void**)&d_b, n * size);
    hipMalloc((void**)&d_c, n * size);

    int a[n], b[n], c[n];
    for (int i = 0; i < n; i++) {
        a[i] = i;        // a[i] = 0, 1, 2, ..., 999
        b[i] = i;        // b[i] = 0, 1, 2, ..., 999
    }

    hipMemcpy(d_a, a, n * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;  // Equivalent to ceil(n / 256)
    vector_addition<<<grid_size, block_size>>>(d_a, d_b, d_c);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipMemcpy(c, d_c, n * size, hipMemcpyDeviceToHost);

    printf("array after vector addition is: \n");
    for (int i = 0; i < 20; i++) {  
        printf("%d ", c[i]);
    }
    printf("...\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}