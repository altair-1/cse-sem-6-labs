#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>  

#define PI 3.14159265359

const int n = 6;

__global__ void angles_to_sine(double *a, double *b) {
    int tid = threadIdx.x;
    if (tid < n) {
        b[tid] = sin(a[tid]);  
    }
}

int main() {
    double *d_a, *d_b; // device copies
    int size = sizeof(double);

    hipMalloc((void**)&d_a, n * size);
    hipMalloc((void**)&d_b, n * size);

    double a[n] = {0, PI / 6, PI / 4, PI / 3, PI / 2, PI};
    double b[n];

    hipMemcpy(d_a, a, n * size, hipMemcpyHostToDevice);

    angles_to_sine<<<1, n>>>(d_a, d_b);

    hipMemcpy(b, d_b, n * size, hipMemcpyDeviceToHost);

    printf("1D array containing sine of angles is: \n");
    for (int i = 0; i < n; i++) {
        printf("sin(%.2lf) = %.2lf \n", a[i], b[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    return 0;
}