#include "hip/hip_runtime.h"

#include <stdio.h>

//each thread calculates the final position of its element by counting how many elements are less than it
__global__ void parallelSelectionSort(int*d_a,int*d_c,int s){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int k=0;
    for(int j=0;j<s;j++){
        if((d_a[j]<d_a[i])||(d_a[j]==d_a[i]&&j>i))
            k++;
    }
    d_c[k]=d_a[i];
}

int main(){
    int n;
    printf("enter number of elements: ");
    scanf("%d",&n);
    int a[n],c[n];
    int *d_a,*d_c;

    hipMalloc((void **)&d_a,n*sizeof(int));
    hipMalloc((void **)&d_c,n*sizeof(int));

    printf("enter the elements: \n");
    for(int i=0;i<n;i++) scanf("%d",&a[i]);
    
    hipMemcpy(d_a,a,n*sizeof(int),hipMemcpyHostToDevice);
    parallelSelectionSort<<<1, n>>>(d_a,d_c,n);
    hipMemcpy(c,d_c,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("sorted array: \n");
    for(int i=0;i<n;i++) 
        printf("%d ",c[i]);
        printf("\n"); 
    hipFree(d_a);
    hipFree(d_c);
}