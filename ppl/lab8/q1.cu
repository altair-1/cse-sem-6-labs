#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void addMatrixByRow(int *a, int *b, int *c, int m, int n){
    int row = threadIdx.x;
    if(row < m){
        for(int col = 0; col < n; col++){
            c[row * n + col] = a[row * n + col] + b[row * n + col];
        }
    }
}

__global__ void addMatrixByColumn(int *a, int *b, int *c, int m, int n){
    int col = threadIdx.x;
    if(col < n){
        for(int row = 0; row < m; row++){
            c[row * n + col] = a[row * n + col] + b[row * n + col];
        }
    }
}

__global__ void addMatrixByElement(int *a, int *b, int *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;    
    if (row < m && col < n) {
        c[row * n + col] = a[row * n + col] + b[row * n + col];
    }
}

void printMatrix(int *m, int m_rows, int n_cols){
    for(int i = 0; i < m_rows; i++){
        for(int j = 0; j < n_cols; j++){
            printf("%d\t", m[i*n_cols + j]);
        }
        printf("\n");
    }
}

int main(){
    int *h_a, *h_b, *h_c, m, n, *d_a, *d_b, *d_c;
    printf("enter the number of rows (m) of the matrices: ");
    scanf("%d", &m);
    printf("enter the number of columns (n) of the matrices: ");
    scanf("%d", &n);
    h_a = (int*)malloc(m * n * sizeof(int));
    h_b = (int*)malloc(m * n * sizeof(int));
    h_c = (int*)malloc(m * n * sizeof(int));
    printf("\n");
    printf("enter the elements for matrix A: \n");
    for(int i = 0; i < m*n; i++) scanf("%d", &h_a[i]);
    printf("\n");
    printf("enter the elements for matrix B: \n");
    for(int i = 0; i < m*n; i++) scanf("%d", &h_b[i]);
    printf("\n");
    hipMalloc((void**)&d_a, m * n * sizeof(int));
    hipMalloc((void**)&d_b, m * n * sizeof(int));
    hipMalloc((void**)&d_c, m * n * sizeof(int));
    hipMemcpy(d_a, h_a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, m * n * sizeof(int), hipMemcpyHostToDevice);
    // row-wise computation
    addMatrixByRow<<<1, m>>>(d_a, d_b, d_c, m, n);
    hipMemcpy(h_c, d_c, m * n * sizeof(int), hipMemcpyDeviceToHost);
    printf("resultant matrix after row-wise computation: \n");
    printMatrix(h_c, m, n);
    printf("\n");
    // column-wise computation
    addMatrixByColumn<<<1, n>>>(d_a, d_b, d_c, m, n);
    hipMemcpy(h_c, d_c, m * n * sizeof(int), hipMemcpyDeviceToHost);
    printf("resultant matrix after column-wise computation: \n");
    printMatrix(h_c, m, n);
    printf("\n");
    // element-wise computation
    dim3 threadsPerBlock(16, 16); 
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);                   
    addMatrixByElement<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n);
    hipMemcpy(h_c, d_c, m * n * sizeof(int), hipMemcpyDeviceToHost);
    printf("resultant matrix after element-wise computation: \n");
    printMatrix(h_c, m, n);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}